
#include <hip/hip_runtime.h>
﻿//#include "../../../include/PointCloudMatch/GPUCPDMatch/GPUCPDMatch.cuh"
//#include <cusolverDn.h>
//#include <fstream>
//#include <iostream>
//
////G矩阵的逆=====================================================================
//void CalGMatInvKernel(float* pA, int M, float* eigenVal)
//{
//	cusolverDnHandle_t handle;
//	cusolverDnCreate(&handle);
//	int info_gpu = 0;
//
//	float* d_A = NULL; cudaMalloc((void**)&d_A, sizeof(float) * M * M);
//	float* d_W = NULL; cudaMalloc((void**)&d_W, sizeof(float) * M);
//	int* devInfo = NULL; cudaMalloc((void**)&devInfo, sizeof(int));
//	cudaMemcpy(d_A, pA, sizeof(float) * M * M, cudaMemcpyHostToDevice);//数据从主机端传至设备端
//
//	float* d_work = NULL;
//	int lwork = 0;
//	cusolverEigMode_t jobz = CUSOLVER_EIG_MODE_VECTOR; 
//	cublasFillMode_t uplo = CUBLAS_FILL_MODE_UPPER;  //由于是对称的，这里采用上三角
//
//	//计算GPU空间
//	cusolverDnSsyevd_bufferSize(handle, jobz, uplo, M, d_A, M, d_W, &lwork);//计算evd计算所需存储空间,保存到lwork中
//	cudaMalloc((void**)&d_work, sizeof(float) * lwork);
//
//	//特征分解
//	cusolverDnSsyevd(handle, jobz, uplo, M, d_A, M, d_W, d_work, lwork, devInfo);
//	cudaDeviceSynchronize();
//
//	//数据传回主机
//	cudaMemcpy(pA, d_A, sizeof(float) * M * M, cudaMemcpyDeviceToHost);
//	cudaMemcpy(eigenVal, d_W, sizeof(float) * M, cudaMemcpyDeviceToHost);
//	cudaMemcpy(&info_gpu, devInfo, sizeof(int), cudaMemcpyDeviceToHost);
//	cusolverDnDestroy(handle);
//}
////==============================================================================

////求矩阵的逆====================================================================
//void GPUCalMatInv(float* pA, int M, float* pC)
//{
//	cublasHandle_t handle;
//	cublasCreate_v2(&handle);
//
//	int* INFO;
//	int* P;
//	cudaMalloc((void**)&INFO, sizeof(int));
//	cudaMalloc((void**)&P, sizeof(int));
//
//	float* d_A;
//	float* d_C;
//	cudaMalloc((void**)&d_A, M * M * sizeof(float));
//	cudaMalloc((void**)&d_C, M * M * sizeof(float));
//	cudaMemcpy(d_A, pA, M * M * sizeof(float), cudaMemcpyHostToDevice);
//
//	float** A = (float**)malloc(sizeof(float*));
//	float** A_d;
//	cudaMalloc((void**)&A_d, sizeof(float*));
//	A[0] = d_A;
//	cudaMemcpy(A_d, A, sizeof(float*), cudaMemcpyHostToDevice);
//
//	//LU分解
//	cublasSgetrfBatched(handle, M, A_d, M, P, INFO, 1);
//	int* INFOh = new int[1];
//	cudaMemcpy(INFOh, INFO, sizeof(int), cudaMemcpyDeviceToHost);
//
//	float* C[1];
//	float** C_d;
//	cudaMalloc((void**)&C_d, sizeof(float*));
//	C[0] = d_C;
//	cudaMemcpy(C_d, C, sizeof(float*), cudaMemcpyHostToDevice);
//	cublasSgetriBatched(handle, M, A_d, M, P, C_d, M, INFO, 1);
//
//	cudaMemcpy(INFOh, INFO, sizeof(int), cudaMemcpyDeviceToHost);
//	cudaMemcpy(pC, d_C, M * M * sizeof(float), cudaMemcpyDeviceToHost);
//	cudaFree(A_d); free(A);
//	cudaFree(C_d);
//	cudaFree(INFO); 
//	cudaFree(d_A);
//	cudaFree(d_C);
//	cublasDestroy_v2(handle); 
//	if (INFOh != nullptr)
//	{
//		delete[] INFOh;
//		INFOh = nullptr;
//	}
//}
////==============================================================================
//
////矩阵相乘======================================================================
//void GPUCalMatMul(float* pA, float* pB, float *pC, int m, int n, int k)
//{
//	cublasHandle_t handle;
//	cublasCreate_v2(&handle);
//	cublasOperation_t tranpose = CUBLAS_OP_N;
//
//	float* d_A;
//	float* d_B;
//	float* d_C;
//	cudaMalloc((void**)&d_A, m * k * sizeof(float));
//	cudaMalloc((void**)&d_B, k * n * sizeof(float));
//	cudaMalloc((void**)&d_C, m * n * sizeof(float));
//	cudaMemcpy(d_A, pA, m * k * sizeof(float), cudaMemcpyHostToDevice);
//	cudaMemcpy(d_B, pB, k * n * sizeof(float), cudaMemcpyHostToDevice);
//
//	const float beta[1] = { 0.0f };
//	const float alpha[1] = { 1.0f };
//	cublasSgemm_v2(handle, tranpose, tranpose, m, n, k, alpha, d_A, m, d_B, k, beta, d_C, m);
//	cudaMemcpy(pC, d_C, m * n * sizeof(float), cudaMemcpyDeviceToHost);
//
//	cublasDestroy_v2(handle);
//	cudaFree(d_A);
//	cudaFree(d_B);
//	cudaFree(d_C);
//}
////==============================================================================
